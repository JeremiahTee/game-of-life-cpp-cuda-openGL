#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <iostream>

typedef int cell_arr[500];

__global__
void kernel_computeMedecineCells(int radius, int x, int y, cell_arr* cell, int medCount)
{
	for (uint32_t i = 0; i < 500 - 5; i++)
	{
		for (uint32_t j = 0; j < 500 - 5; j++)
		{
			uint32_t a = i - x;
			uint32_t b = j - y;

			//The cell at x,y is inside the circle
			if (a * a + b * b <= radius * radius)
			{
				cell[i][j] = 2;
			}
		}
	}
}

void spawnMedecineCellsCUDA(int radius, int x, int y, int cell[500][500], int medCount)
{
	cell_arr* d_cell;
	size_t size = 500 * 500 * sizeof(int);
	
	//Allocate memory on device
	hipMalloc(&d_cell, size);

	//Copy cell array from host to the device
	hipMemcpy(d_cell, cell, size, hipMemcpyHostToDevice);

	//Run kernel
	kernel_computeMedecineCells << <1,1 >> >(radius, x, y, d_cell, medCount);
	
	// Wait on GPU before accessing host
	hipDeviceSynchronize();

	//Copy cell array from device to the host
	hipMemcpy(cell, d_cell, size, hipMemcpyDeviceToHost);

	//Free allocated memory
	hipFree(d_cell);
}